
#include <hip/hip_runtime.h>
__global__ void SMWatSSM(float* SSMA, float* SSMB, float* CSM, int M, int N, int diagLen, int diagLenPow2, float hvPenalty, int flip) {
    //Have circularly rotating system of 3 buffers
    extern __shared__ float x[]; //Circular buffer
    int off = 0;
    int upoff = 0;

    //Other local variables
    int i, k;
    int i1, i2, j1, j2;
    int thisi, thisj;
    int idx;
    float val, score;
    int ci = blockIdx.x;
    int cj = blockIdx.y;
    int finished = 0;


    //Figure out K (number of batches)
    int K = diagLenPow2 >> 9;
    if (K == 0) {
        K = 1;
    }

    //Initialize all buffer elements to -1
    for (k = 0; k < K; k++) {
        for (off = 0; off < 3; off++) {
            if (512*k + threadIdx.x < diagLen) {
                x[512*k + threadIdx.x + off*diagLen] = -1;
            }
        }
    }
    off = 0;

    //Process each diagonal
    for (i = 0; i < N + M - 1; i++) {
        if (finished) {
            break;
        }
        //Figure out the bounds of this diagonal
        i1 = i;
        j1 = 0;
        upoff = -1;
        if (i1 >= M) {
            i1 = M-1;
            j1 = i - (M-1);
            upoff = 0;
        }
        j2 = i;
        i2 = 0;
        if (j2 >= N) {
            j2 = N-1;
            i2 = i - (N-1);
        }
        //Update each batch
        for (k = 0; k < K; k++) {
            idx = k*512 + threadIdx.x;
            if (idx >= diagLen) {
                break;
            }
            thisi = i1 - idx;
            thisj = j1 + idx;
            if (thisi < i2 || thisj > j2) {
                x[off*diagLen + idx] = -1;
                continue;
            }
            if (flip) {
                val = SSMA[(M-ci)*M + (M-thisi)] - SSMB[(N-cj)*N + N-thisj];
            }
            else {
                val = SSMA[ci*M + thisi] - SSMB[cj*N + thisj];
            }
            if (val < 0) {
                val = val*-1.0f;
            }
            val = expf(-val/0.09f)-0.6f;
            score = 0.0f;
            if (thisi == 0 && thisj == 0 || thisi == 1 && thisj == 1) {
                score = val;
            }
            else if (thisi == 0 || thisj == 0) {
                score = val + hvPenalty;
            }
            if (score < 0.0f) {
                score = 0.0f;
            }
            //Above
            if (idx + upoff + 1 < N + M - 1 && thisi > 0) {
                if (x[((off+1)%3)*diagLen + idx + upoff + 1] > -1) {
                    if (score < val + x[((off+1)%3)*diagLen + idx + upoff + 1] + hvPenalty) {
                        score = val + x[((off+1)%3)*diagLen + idx + upoff + 1] + hvPenalty;
                    }
                }
            }
            if (idx + upoff >= 0 && thisj > 0) {
                //Left
                if (x[((off+1)%3)*diagLen + idx + upoff] > -1) {
                    if (score < x[((off+1)%3)*diagLen + idx + upoff] + val + hvPenalty) {
                        score = x[((off+1)%3)*diagLen + idx + upoff] + val + hvPenalty;
                    }
                }
            }
            if (i1 == M-1 && j1 > 1) {
                upoff = 1;
            }
            if (idx + upoff >= 0 && thisi > 0) {
                //Diagonal
                if (x[((off+2)%3)*diagLen + idx + upoff] > -1) {
                    if (score < x[((off+2)%3)*diagLen + idx + upoff] + val < score) {
                        score = x[((off+2)%3)*diagLen + idx + upoff] + val;
                    }
                }
            }
            x[off*diagLen + idx] = score;
            if (thisi == ci && thisj == cj) {
                CSM[ci*N + cj] = score;
                finished = 1;
            }
        }
        off = (off + 2) % 3; //Cycle buffers
        __syncthreads();
    }
}
